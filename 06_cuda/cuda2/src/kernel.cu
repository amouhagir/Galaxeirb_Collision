#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "header.h"


__global__ void kernel_update_pos( particule_t *p, vector_t *acc, int size ) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if ( i < size)
	{
		p[i].vx += acc[i].x*MASSFACTOR*DAMP;
		p[i].vy += acc[i].y*MASSFACTOR*DAMP;
		p[i].vz += acc[i].z*MASSFACTOR*DAMP;
		p[i].x += (p[i].vx)*DT;
		p[i].y += (p[i].vy)*DT;
		p[i].z += (p[i].vz)*DT;
	}

}

// 1. utilisation des builtins -> 600 fps
// float3 (x, y, z)
// float4 (x, y, z, w)
// x, y, z, vx, vy, vz, m
// float3, float4

// 2. utilisation de la mémoire partagée -> 1200 fps
// mémoire partagée entre thread d'un bloc
// (NVIDIA N-BODY GPU GEM)

// 3. utilisation de la pinned memory -> 1400-1500 fps

__global__ void kernel_update_acc( particule_t *p, vector_t *acc, int size ) {
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	//int stride = gridDim.x * blockDim.x;
	int i=0;

	float dx,dy,dz,d,fact;
	if ( j < size ) {
	//for(j=index;j<size; j+= stride){
		acc[j].x = 0.0f;
		acc[j].y = 0.0f;
		acc[j].z = 0.0f;
		float xj = p[j].x;
		float yj = p[j].y;
		float zj = p[j].z;


		for (i = 0; i < size; ++i)
		{

			dx = p[i].x-xj;
			dy = p[i].y-yj;
			dz = p[i].z-zj;

			d  = dx*dx+dy*dy+dz*dz;
			if ( d < 1.0 ) d = 1.0;
			fact=p[i].m/(d*sqrtf(d));
			acc[j].x += dx*fact;
			acc[j].y += dy*fact;
			acc[j].z += dz*fact;

		}

	}
}

void update_acc( int nblocks, int nthreads, particule_t *p, vector_t *acc, int size) {
	kernel_update_acc<<<nblocks, nthreads>>>( p, acc, size);

}



void update_position( int nblocks, int nthreads, particule_t *p, vector_t *acc, int size) {
	kernel_update_acc<<<nblocks, nthreads>>>( p, acc, size);
	//hipDeviceSynchronize();
	kernel_update_pos<<<nblocks, nthreads>>>( p, acc, size);
}
